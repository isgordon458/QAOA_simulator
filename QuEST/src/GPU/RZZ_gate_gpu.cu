#include "hip/hip_runtime.h"
#include <QuEST.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>

__managed__ qreal H;

void initH(Qureg qureg) {
    int numQubits = qureg.numQubitsInStateVec;
    H = pow(1./sqrt(2.), numQubits);
}

ull* init1DGraph(Qureg qureg) {
   ull *graph;
   hipMallocManaged(&graph, qureg.numQubitsInStateVec * sizeof(ull));
   memset(graph, 0, qureg.numQubitsInStateVec * sizeof(ull));
   return graph;
}

void free1DGraph(ull* graph) {
    assert(hipFree(graph) == hipSuccess);
}

void addEdgeTo1DGraph(Qureg qureg, ull* graph, int qubit1, int qubit2) {
    int numQubits = qureg.numQubitsInStateVec;
    assert(qubit1 < numQubits);
    assert(qubit2 < numQubits);
    if (qubit1 > qubit2) {
        int tmp = qubit1;
        qubit1 = qubit2;
        qubit2 = tmp;
    }
    graph[qubit1] |= 1ull << qubit2;
}

bool* init2DGraph(Qureg qureg) {
    int numQubits = qureg.numQubitsInStateVec;
    bool *graph;
    assert(hipMallocManaged(&graph, numQubits*numQubits * sizeof(bool)) == hipSuccess);
    return graph;
}

qreal* initWeights(Qureg qureg) {
    int numQubits = qureg.numQubitsInStateVec;
    qreal *weights;
    assert(hipMallocManaged(&weights, numQubits*numQubits * sizeof(qreal)) == hipSuccess);
    return weights;
}

void addEdgeTo2DGraph(Qureg qureg, bool* graph, qreal *weights, int qubit1, int qubit2, qreal weight) {
    int numQubits = qureg.numQubitsInStateVec;
    assert(qubit1 < numQubits);
    assert(qubit2 < numQubits);
    if (qubit1 > qubit2) {
        int tmp = qubit1;
        qubit1 = qubit2;
        qubit2 = tmp;
    }
    graph[qubit1*numQubits+qubit2] = 1;
    weights[qubit1*numQubits+qubit2] = weight;
}

void free2DGraph(bool* graph) {
    assert(hipFree(graph) == hipSuccess);
}

void freeWeights(qreal *weights)
{
    assert(hipFree(weights) == hipSuccess);
}

__global__
void _rotationCompressionWeighted(qreal *real, qreal *imag, int numQubits, qreal angle, bool *graph, qreal *weights, int numGates, bool isFirstLayer){

    ull tidx = blockIdx.x * blockDim.x + threadIdx.x;
    ull b = tidx;

    qreal sum = 0;
    // count number of 1
    for (int i = 0; i < numQubits; i++) {
        for (int j = i + 1; j < numQubits; j++) {
            int idx = i*numQubits+j;
            if (graph[idx]) {
                if (((b>>i)&0b1) ^ ((b>>j)&0b1))
                    sum += weights[idx];
                else
                    sum -= weights[idx];
            }
        }
    }

    qreal contracted_rzz[2];
    sincos(sum*angle/2, &contracted_rzz[1], &contracted_rzz[0]);

    if (isFirstLayer) {
        real[tidx] = contracted_rzz[0] * H;
        imag[tidx] = contracted_rzz[1] * H;
    } else {
        qreal in[2];
        in[0] = real[tidx];
        in[1] = imag[tidx];
        real[tidx] = in[0] * contracted_rzz[0] - in[1] * contracted_rzz[1];
        imag[tidx] = in[0] * contracted_rzz[1] + in[1] * contracted_rzz[0];
    }
}

void rotationCompressionWeighted(Qureg qureg, qreal angle, bool *graph, qreal *weights, bool isFirstLayer)
{
    int numQubits = qureg.numQubitsInStateVec;
    ull numAmps = 1ull << numQubits;

    ull grid = 1;
    ull block = 128;

    if (numAmps > block)
        grid = numAmps / 128;
    else
        block = numAmps;

    int numGates = 0;
    for (int i = 0; i < numQubits; i++) {
        for (int j = i + 1; j < numQubits; j++) {
            numGates++;
        }
    }

    _rotationCompressionWeighted<<<grid, block>>>(qureg.deviceStateVec.real, qureg.deviceStateVec.imag, qureg.numQubitsInStateVec, angle, graph, weights, numGates, isFirstLayer);
}

__global__
void _rotationCompressionUnweighted(qreal *real, qreal *imag, int numQubits, qreal angle, ull *graph, int numGates, bool isFirstLayer) {

    ull tidx = blockIdx.x * blockDim.x + threadIdx.x;
    ull b = tidx;
    
    int Cp = 0;
    // count number of 1
    for (int i = 0; i < numQubits; i++) {
        ull strxor;
        ull msk = -((b >> i) & 1LL);
        strxor = graph[i] & (b ^ msk);
        Cp += __popcll(strxor);
    }

    qreal contracted_rzz[2];
    sincos((2*Cp-numGates)*angle/2, &contracted_rzz[1], &contracted_rzz[0]);

    if (isFirstLayer) {
        real[tidx] = contracted_rzz[0] * H;
	    imag[tidx] = contracted_rzz[1] * H;
    } else {
        qreal in[2];
        in[0] = real[tidx];
        in[1] = imag[tidx];
        real[tidx] = in[0] * contracted_rzz[0] - in[1] * contracted_rzz[1];
        imag[tidx] = in[0] * contracted_rzz[1] + in[1] * contracted_rzz[0];
    }
}

void rotationCompressionUnweighted(Qureg qureg, qreal angle, ull *graph, bool isFirstLayer) {

    int numQubits = qureg.numQubitsInStateVec;
    ull numAmps = 1ull << numQubits;

    ull grid = 1;
    ull block = 128;

    if (numAmps > block)
        grid = numAmps / 128;
    else
        block = numAmps;

    int numGates = 0;
    for (int i = 0; i < numQubits; i++)
        numGates += __builtin_popcountll(graph[i]);

    _rotationCompressionUnweighted<<<grid, block>>>(qureg.deviceStateVec.real, qureg.deviceStateVec.imag, qureg.numQubitsInStateVec, angle, graph, numGates, isFirstLayer);
}